#include "hip/hip_runtime.h"
/* This code will generate a fractal image. Uses OpenCV, to compile:
   nvcc Cuda.cu `pkg-config --cflags --libs opencv`  */
   #include <stdio.h>
   #include <stdlib.h>
   #include <opencv/highgui.h>
   #include "utils/cheader.h"
   
   
   typedef enum color {BLUE, GREEN, RED} Color;
   
   __global__ void convert_to_hsv(unsigned char *src, float *hsv, int width, int heigth, int step, int channels) {
       float r, g, b;
       float h, s, v;
       int ren,col;
   
       ren = blockIdx.x;
       col = threadIdx.x;
   
       r = src[(ren * step) + (col * channels) + RED] / 255.0f;
       g = src[(ren * step) + (col * channels) + GREEN] / 255.0f;
       b = src[(ren * step) + (col * channels) + BLUE] / 255.0f;
       
       float max = fmax(r, fmax(g, b));
       float min = fmin(r, fmin(g, b));
       float diff = max - min;
       
       v = max;
       
       if(v == 0.0f) { // black
           h = s = 0.0f;
       } else {
           s = diff / v;
           if(diff < 0.001f) { // grey
               h = 0.0f;
           } else { // color
               if(max == r) {
                   h = 60.0f * (g - b)/diff;
                   if(h < 0.0f) { h += 360.0f; }
               } else if(max == g) {
                   h = 60.0f * (2 + (b - r)/diff);
               } else {
                   h = 60.0f * (4 + (r - g)/diff);
               }
           }		
       }
   
       float minh=40.0f;
       float maxh=200.0f;
       // if to check the color blindness line, if the pixel is in this line i change the color to other color base shifting the h
       
       float minis = 0;
       float maxs = 100;
       float miniv = 0;
       float maxv = 100;
           
   
       if (h > minh && h < maxh && s > minis && s < maxs && v > miniv && v < maxv){
           
           hsv[(ren * step) + (col * channels) + RED] =  (float) (h + 140.0f);
           hsv[(ren * step) + (col * channels) + GREEN] = (float) (s);
           hsv[(ren * step) + (col * channels) + BLUE] = (float) (v);
       } else { // this keep the pixel if it is out of the color blindnessline
           hsv[(ren * step) + (col * channels) + RED] =  (float) (h);
           hsv[(ren * step) + (col * channels) + GREEN] = (float) (s);
           hsv[(ren * step) + (col * channels) + BLUE] = (float) (v);
       }
       
       
   }
   
   __global__ void convert_to_rgb(float *hsv, unsigned char *dest, int width, int heigth, int step, int channels) {
       float r, g, b;
       float h, s, v;
       int ren,col;
   
       ren = blockIdx.x;
       col = threadIdx.x;	
       h = hsv[(ren * step) + (col * channels) + RED];
       s = hsv[(ren * step) + (col * channels) + GREEN];
       v = hsv[(ren * step) + (col * channels) + BLUE];
       
       float f = h/60.0f;
       float hi = floorf(f);
       f = f - hi;
       float p = v * (1 - s);
       float q = v * (1 - s * f);
       float t = v * (1 - s * (1 - f));
       
       if(hi == 0.0f || hi == 6.0f) {
           r = v;
           g = t;
           b = p;
       } else if(hi == 1.0f) {
           r = q;
           g = v;
           b = p;
       } else if(hi == 2.0f) {
           r = p;
           g = v;
           b = t;
       } else if(hi == 3.0f) {
           r = p;
           g = q;
           b = v;
       } else if(hi == 4.0f) {
           r = t;
           g = p;
           b = v;
       } else {
           r = v;
           g = p;
           b = q;
       }
   
       dest[(ren * step) + (col * channels) + RED] =  (unsigned char) __float2uint_rn(255.0f * r);
       dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) __float2uint_rn(255.0f * g);
       dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) __float2uint_rn(255.0f * b);
   }
   
   void compute_pixel(IplImage *src, IplImage *dest , int ren, int col) {
       int step;
       float r, g, b;
       float h, s, v;
       
       step = src->widthStep / sizeof(uchar);
       
       r = (float) (src->imageData[(ren * step) + (col * src->nChannels) + RED] / 255.0f);
       g = (float) (src->imageData[(ren * step) + (col * src->nChannels) + GREEN] / 255.0f);
       b = (float) (src->imageData[(ren * step) + (col * src->nChannels) + BLUE] / 255.0f);
       
       float max = fmax(r, fmax(g, b));
       float min = fmin(r, fmin(g, b));
       float diff = max - min;
       v = max;
       
       if(v == 0.0f) { 		// black
           h = s = 0.0f;
       } else {
           s = diff / v;
           if(diff < 0.001f) { // grey
               h = 0.0f;
           } else { 			// color
               if(max == r) {
                   h = 60.0f * (g - b)/diff;
                   if(h < 0.0f) { h += 360.0f; }
                   } else if(max == g) {
                       h = 60.0f * (2 + (b - r) / diff);
                   } else {
                       h = 60.0f * (4 + (r - g) / diff);
                   }
           }		
               
   
           
       }
   
       
       // if to check the color blindness line, if the pixel is in this line i change the color to other color base shifting the h
       float minh=40.0f;
       float maxh=160.0f;
   /*
       float minis = 0.0f;
       float maxs = 100.0f;
       float miniv = 0.0f;
       float maxv = 100.0f;
       */	
   
       if (h > minh && h < maxh ){
           
           h = (h+140.0f);
           s = s ;
           v = v;
       } else { // this keep the pixel if it is out of the color blindnessline
           h = h;
           s = s;
           v = v;		
       }
       
       float f = h/60.0f;
       float hi = floorf(f);
       f = f - hi;
       float p = v * (1 - s);
       float q = v * (1 - s * f);
       float t = v * (1 - s * (1 - f));
       
       if(hi == 0.0f || hi == 6.0f) {
           r = v;
           g = t;
           b = p;
       } else if(hi == 1.0f) {
           r = q;
           g = v;
           b = p;
       } else if(hi == 2.0f) {
           r = p;
           g = v;
           b = t;
       } else if(hi == 3.0f) {
           r = p;
           g = q;
           b = v;
       } else if(hi == 4.0f) {
           r = t;
           g = p;
           b = v;
       } else {
           r = v;
           g = p;
           b = q;
       }
       dest->imageData[(ren * step) + (col * dest->nChannels) + RED] =  (unsigned char) (r * 255.0f );
       dest->imageData[(ren * step) + (col * dest->nChannels) + GREEN] = (unsigned char) (g * 255.0f );
       dest->imageData[(ren * step) + (col * dest->nChannels) + BLUE] = (unsigned char) (b * 255.0f);
   }
       
   void compute_image_seq(IplImage *src, IplImage *dest) {
       int index, size;
       int ren, col;
       
       size = src->width * src->height;
       for (index = 0; index < size; index++) {
           ren = index / src->width;
           col = index % src->width;
           compute_pixel(src, dest, ren, col);
       }
   }
   
   
   int main(int argc, char* argv[]) {
       int size, step, size2;
       int i;
       double acum, acum2; 
       float *dev_hsv;
       unsigned char *dev_src ,*dev_dest;
       
           
       if (argc != 2) {
           printf("usage: %s source_file\n", argv[0]);
           return -1;
       }
       
       IplImage *src = cvLoadImage(argv[1], CV_LOAD_IMAGE_COLOR);
       IplImage *hsv = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_32F, 3);
       IplImage *dest = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);
       IplImage *dest2 = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);
   
       
       if (!src) {
           printf("Could not load image file: %s\n", argv[1]);
           return -1;
       }
       
       size = src->width * src->height * src->nChannels * sizeof(uchar);
       size2 = src->width * src->height * src->nChannels * sizeof(float);
       hipMalloc((void**) &dev_src, size);
       hipMalloc((void**) &dev_hsv, size2);
       hipMalloc((void**) &dev_dest, size);
       
       acum = 0;
       step = src->widthStep / sizeof(uchar);
       printf("Starting Sequential...\n");
       acum2 = 0;
       for (i = 0; i < N; i++) {
           start_timer();
           compute_image_seq(src, dest2);
           acum2 += stop_timer();
       }
       printf("avg time Sequential = %.5lf ms\n", (acum2 / N));
   
       printf("Starting MultiThreading...\n");
       for (i = 0; i < N; i++) {
           start_timer();
           hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);
           convert_to_hsv<<<src->height, src->width>>>(dev_src, dev_hsv, src->width, src->height, step, src->nChannels);
           hipMemcpy(hsv->imageData, dev_hsv, size2, hipMemcpyDeviceToHost);
   
           convert_to_rgb<<<src->height, src->width>>>(dev_hsv, dev_dest, src->width, src->height, step, src->nChannels);
   
           hipMemcpy(dest->imageData, dev_dest, size, hipMemcpyDeviceToHost);
           acum += stop_timer();
       }
   
   
   
       hipFree(dev_dest);
       hipFree(dev_hsv);
       hipFree(dev_src);
       
       printf("avg time MultiThreading = %.5lf ms\n", (acum / N));
   
       
       cvShowImage("Image (Original)", src);
       cvShowImage("Image (Final)", dest);
       cvWaitKey(0);
       cvDestroyWindow("Image (Original)");
       cvDestroyWindow("Image (Final)");
   
       return 0;
   }
   